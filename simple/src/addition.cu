
#include <hip/hip_runtime.h>
#include <stdio.h>

/* cuda kernel declared and defined */
__global__ void add( int a, int b, int *c ){
	*c = a + b;
}

int main( void ) {
	int c;
	int *dev_c;
	/* allocates memory on the device */
	hipMalloc( (void**)&dev_c, sizeof(int));

	/*call to kernel*/
	add<<<1,1>>>(2, 7, dev_c);	
	
	/* copies dev_c into c */	
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2 + 7 = %d\n", c);

	hipFree(dev_c);

	return 0;

}
